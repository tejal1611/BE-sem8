#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define BLOCK_SIZE 2
__global__ void gpuMM(float *A, float *B, float *C, int N)
{
// Matrix multiplication for NxN matrices C=A*B
// Each thread computes a single element of C
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;
float sum = 0.f;
for (int n = 0; n < N; ++n)
sum += A[row*N+n]*B[n*N+col];
C[row*N+col] = sum;
}
int main(int argc, char *argv[])
{int N;float K;
// Perform matrix multiplication C = A*B
// where A, B and C are NxN matrices
// Restricted to matrices where N = K*BLOCK_SIZE;
cout<<"Enter a Value for Size/2 of matrix";
cin>>K;
K = 1;
N = K*BLOCK_SIZE;
cout << "\n Executing Matrix Multiplcation" << endl;
cout << "\n Matrix size: " << N << "x" << N << endl;
// Allocate memory on the host
float *hA,*hB,*hC;
hA = new float[N*N];
hB = new float[N*N];
hC = new float[N*N];
// Initialize matrices on the host
for (int j=0; j<N; j++){
for (int i=0; i<N; i++){
hA[j*N+i] = 2;
hB[j*N+i] = 4;
}
}
// Allocate memory on the device
int size = N*N*sizeof(float); // Size of the memory in bytes
float *dA,*dB,*dC;
hipMalloc(&dA,size);
hipMalloc(&dB,size);
hipMalloc(&dC,size);
dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
dim3 grid(K,K);
cout<<"\n Input Matrix 1 \n";
for (int row=0; row<N; row++){
for (int col=0; col<N; col++){
cout<<hA[row*col]<<" ";
}
cout<<endl;
}
cout<<"\n Input Matrix 2 \n";
for (int row=0; row<N; row++){
for (int col=0; col<N; col++){
cout<<hB[row*col]<<" ";
}
cout<<endl;
}
// Copy matrices from the host to device
hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
//Execute the matrix multiplication kernel
gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
// Now do the matrix multiplication on the CPU
/*float sum;
for (int row=0; row<N; row++){
for (int col=0; col<N; col++){
sum = 0.f;
for (int n=0; n<N; n++){
sum += hA[row*N+n]*hB[n*N+col];
}
hC[row*N+col] = sum;
cout << sum <<" ";
}
cout<<endl;
}*/
// Allocate memory to store the GPU answer on the host
float *C;
C = new float[N*N];
// Now copy the GPU result back to CPU
hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);
// Check the result and make sure it is correct
cout <<"\n\n\n\n\n Resultant matrix\n\n";
for (int row=0; row<N; row++){
for (int col=0; col<N; col++){
cout<<C[row*col]<<" ";
}
cout<<endl;
}
cout << "Finished." << endl;
}