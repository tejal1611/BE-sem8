

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <climits>

__global__ void min_reduction_kernel(int* arr, int size, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        atomicMin(result, arr[tid]);

    }
}

__global__ void max_reduction_kernel(int* arr, int size, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        atomicMax(result, arr[tid]);
    }
}

__global__ void sum_reduction_kernel(int* arr, int size, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        atomicAdd(result, arr[tid]);
    }
}

__global__ void average_reduction_kernel(int* arr, int size, int* sum) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        atomicAdd(sum, arr[tid]);
    }
}

int main() {
    std::vector<int> arr = {5, 2, 9, 1, 7, 6, 8, 3, 4};
    int size = arr.size();
    int* d_arr;
    int* d_result;
    int result_min = INT_MAX;
    int result_max = INT_MIN;
    int result_sum = 0;

    // Allocate memory on the device
    hipMalloc(&d_arr, size * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_arr, arr.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result_min, sizeof(int), hipMemcpyHostToDevice);

    // Perform min reduction
    min_reduction_kernel<<<(size + 255) / 256, 256>>>(d_arr, size, d_result);
    hipMemcpy(&result_min, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Minimum value: " << result_min << std::endl;

    // Perform max reduction
    hipMemcpy(d_result, &result_max, sizeof(int), hipMemcpyHostToDevice);
    max_reduction_kernel<<<(size + 255) / 256, 256>>>(d_arr, size, d_result);
    hipMemcpy(&result_max, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Maximum value: " << result_max << std::endl;

    // Perform sum reduction
    hipMemcpy(d_result, &result_sum, sizeof(int), hipMemcpyHostToDevice);
    sum_reduction_kernel<<<(size + 255) / 256, 256>>>(d_arr, size, d_result);
    hipMemcpy(&result_sum, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Sum: " << result_sum << std::endl;

    // Perform average reduction
    hipMemcpy(d_result, &result_sum, sizeof(int), hipMemcpyHostToDevice);
    average_reduction_kernel<<<(size + 255) / 256, 256>>>(d_arr, size, d_result);
    hipMemcpy(&result_sum, d_result, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Average: " << static_cast<double>(result_sum) / size << std::endl;

    // Free device memory
    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}

